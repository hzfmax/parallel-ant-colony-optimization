#include "hip/hip_runtime.h"
#include<iostream>
#include<cstdlib>
#include<pthread.h>
#include<semaphore.h>
#include<time.h>
#include "ant.h"
#include "map3d.h"

#define MAX_STEP 10000
#define THREAD_COUNT 4
#define TOTAL_ANTS 60
#define MAX_HORM_LEFT 100
#define MAP_X 50
#define MAP_Y 50
#define MAP_Z 3
#define HOME_X 11
#define HOME_Y 11
#define HOME_Z 1
#define FOOD_X 39
#define FOOD_Y 39
#define FOOD_Z 1
#define BLOCK_SIZE 512

using namespace std;
int tt=0, ff=0, fh=0;
int counter1,counter2;
sem_t update_barrier;
sem_t barrier1,barrier2;
sem_t mutex1,mutex2;
ant ants[TOTAL_ANTS];
map3d mmap;
double *Md1, *Md2, blue_horm[MAP_X*MAP_Y*MAP_Z], red_horm[MAP_X*MAP_Y*MAP_Z];
int total_block, tpoints;


__global__ void horm_update(double *Md, double decline)
{
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	if(index < MAP_X*MAP_Y*MAP_Z)
        Md[index] = Md[index]*decline;
}

int matrix_3d_to_1d(int x,int y,int z)
{
    return z * MAP_X * MAP_Y + y * MAP_X + x;
}

void layout(const char *file_name, double *horm_array)
{
    FILE *fp;
    fp = fopen(file_name,"a+");
    if(fp!=NULL)
    {
        fprintf(fp, "\n%d*%d*%d\n",MAP_X, MAP_Y, MAP_Z);
        for(int i=0;i<MAP_Z;i++)
        {
            for(int j=0;j<MAP_Y;j++)
            {
                for(int k=0;k<MAP_X;k++)
                {
                    fprintf(fp, "%f",horm_array[matrix_3d_to_1d(k,j,i)]);
                    if(k!=MAP_X-1)
                        fprintf(fp, ",");
                    else
                        fprintf(fp, "\n");
                }
            }
        }
    }
}

void* run_ants(void* data)
{
	double horm0, horm1, horm2, horm3, horm4, horm5, horm6, horm7, horm8, horm9;
	unsigned int i, j, pos_x, pos_y, pos_z;
	unsigned int thread_id = *(unsigned int*) data;
	unsigned int istart = thread_id * TOTAL_ANTS / THREAD_COUNT;
	unsigned int iend = (thread_id + 1) * TOTAL_ANTS / THREAD_COUNT;
	double left_horm;
	if (thread_id == THREAD_COUNT - 1) 
		iend = TOTAL_ANTS;

	while(true){
		// cpu: update horm map
		if(thread_id == 0)
		{
			for(j = 0; j < TOTAL_ANTS; j++)
            {
				pos_x = ants[j].get_x(); 
				pos_y = ants[j].get_y(); 
				pos_z = ants[j].get_z(); 
				left_horm = ants[j].get_horm();
				if(ants[j].get_state() == 0/*blue*/) 
					blue_horm[matrix_3d_to_1d(pos_x,pos_y,pos_z)] += left_horm;
				else if(ants[j].get_state() == 1/*red*/) 
					red_horm[matrix_3d_to_1d(pos_x,pos_y,pos_z)] += left_horm;
				else if(ants[j].get_state() == -1/*init blue*/) 
					blue_horm[matrix_3d_to_1d(pos_x,pos_y,pos_z)] += left_horm;
			}
			
			// release barrier
			for(j = 0; j < THREAD_COUNT - 1; j++){
				sem_post(&update_barrier);
			}
		}
		else{
			sem_wait(&update_barrier);
		}
		
		// cpu: get & compute horm & decide direction
		for(int i=istart; i<iend; i++)
        {
            pos_x = ants[i].get_x(); 
			pos_y = ants[i].get_y(); 
			pos_z = ants[i].get_z();
            if(ants[i].get_state() == 1){ // the ant is in red horm
				horm0 = blue_horm[matrix_3d_to_1d(pos_x,pos_y+1,pos_z)];
				horm1 = blue_horm[matrix_3d_to_1d(pos_x-1,pos_y,pos_z)];
				horm2 = blue_horm[matrix_3d_to_1d(pos_x+1,pos_y,pos_z)];
				horm3 = blue_horm[matrix_3d_to_1d(pos_x,pos_y-1,pos_z)];
				horm4 = blue_horm[matrix_3d_to_1d(pos_x,pos_y,pos_z+1)];
				horm5 = blue_horm[matrix_3d_to_1d(pos_x,pos_y,pos_z-1)];
            }
            else if (ants[i].get_state() == 0){ // the ant is in blue horm
				horm0 = red_horm[matrix_3d_to_1d(pos_x,pos_y+1,pos_z)];
				horm1 = red_horm[matrix_3d_to_1d(pos_x-1,pos_y,pos_z)];
				horm2 = red_horm[matrix_3d_to_1d(pos_x+1,pos_y,pos_z)];
				horm3 = red_horm[matrix_3d_to_1d(pos_x,pos_y-1,pos_z)];
				horm4 = red_horm[matrix_3d_to_1d(pos_x,pos_y,pos_z+1)];
				horm5 = red_horm[matrix_3d_to_1d(pos_x,pos_y,pos_z-1)];
            }
            else if (ants[i].get_state() == -1){ // the ant is in blue horm
				horm0 = -blue_horm[matrix_3d_to_1d(pos_x,pos_y+1,pos_z)]+red_horm[matrix_3d_to_1d(pos_x,pos_y+1,pos_z)];
				horm1 = -blue_horm[matrix_3d_to_1d(pos_x-1,pos_y,pos_z)]+red_horm[matrix_3d_to_1d(pos_x-1,pos_y,pos_z)];
				horm2 = -blue_horm[matrix_3d_to_1d(pos_x+1,pos_y,pos_z)]+red_horm[matrix_3d_to_1d(pos_x+1,pos_y,pos_z)];
				horm3 = -blue_horm[matrix_3d_to_1d(pos_x,pos_y-1,pos_z)]+red_horm[matrix_3d_to_1d(pos_x,pos_y-1,pos_z)];
				horm4 = -blue_horm[matrix_3d_to_1d(pos_x,pos_y,pos_z+1)]+red_horm[matrix_3d_to_1d(pos_x,pos_y,pos_z+1)];
				horm5 = -blue_horm[matrix_3d_to_1d(pos_x,pos_y,pos_z-1)]+red_horm[matrix_3d_to_1d(pos_x,pos_y,pos_z-1)];
            }
            ants[i].set_sight(horm0, horm1, horm2, horm3, horm4, horm5);
        	ants[i].decide_direction(mmap);
		}
		
		// barrier
		sem_wait(&mutex1);
		++counter1;
		if (counter1 == THREAD_COUNT)
		{
			counter1 = 0;
			for(j = 0; j < THREAD_COUNT; j++)
				sem_post(&barrier1);
		}
		sem_post(&mutex1);
		sem_wait(&barrier1);
		
		// cpu: decide direction, gpu: decline horm
		if(thread_id == 0)
        {
            hipMemcpy(blue_horm, Md1, tpoints*sizeof(double), hipMemcpyHostToDevice);
            hipMemcpy(red_horm, Md2, tpoints*sizeof(double), hipMemcpyHostToDevice);
            horm_update<<<total_block, BLOCK_SIZE>>>(Md1, 0.99);
            horm_update<<<total_block, BLOCK_SIZE>>>(Md2, 0.99);
            hipMemcpy(red_horm, Md2, tpoints*sizeof(double), hipMemcpyDeviceToHost);
            hipMemcpy(blue_horm, Md1, tpoints*sizeof(double), hipMemcpyDeviceToHost);
		}

		// barrier
		sem_wait(&mutex2);
		++counter2;
		if (counter2 == THREAD_COUNT)
		{
			counter2 = 0;
			if (++tt%100 == 0)
			{
				ff = 0;
				fh = 0;
				for(j = 0; j < TOTAL_ANTS; j++)
                {
					if(ants[j].get_state()==1) 
						++ff;
					else if(ants[j].get_state()==0) 
						++fh;
				}
				cout << tt << "\t" << TOTAL_ANTS-ff-fh << " : " << ff << " : " << fh <<endl;
				if(tt>=MAX_STEP)
				{
				    remove( "blue.txt" );
				    layout("blue.txt", blue_horm);
					remove( "red.txt" );
				    layout("red.txt", red_horm);
					exit(0);
				}
			}
			for(j = 0; j < THREAD_COUNT; j++)
				sem_post(&barrier2);
		}
		sem_post(&mutex2);
		sem_wait(&barrier2);
	}
}

void init()
{	
	counter1 = 0;
	counter2 = 0;
	sem_init(&update_barrier, 0, 0);
	sem_init(&mutex1, 0, 1);
	sem_init(&barrier1, 0, 0);
	sem_init(&mutex2, 0, 1);
	sem_init(&barrier2, 0, 0);

    tpoints = MAP_X*MAP_Y*MAP_Z;
    hipMalloc((void**) &Md1, tpoints*sizeof(double));
    hipMalloc((void**) &Md2, tpoints*sizeof(double));
    total_block = ((tpoints % BLOCK_SIZE) == 0)? (tpoints/BLOCK_SIZE) : (tpoints/BLOCK_SIZE + 1);

	mmap.load_sample(MAP_X,MAP_Y,MAP_Z);
	
	//set home and food point
	mmap.edit(HOME_X,HOME_Y,HOME_Z,101);
	mmap.edit(FOOD_X,FOOD_Y,FOOD_Z,100);
	
	//initial ants
	srand (time(NULL));
	int offset = rand()/2;
	unsigned int set_seed;
	for(int i=0; i<TOTAL_ANTS; ++i)
	{
		ants[i].set_position(HOME_X,HOME_Y,HOME_Z);
		ants[i].set_home_xyz(HOME_X,HOME_Y,HOME_Z);
		
		ants[i].set_horm(MAX_HORM_LEFT);
		ants[i].set_max_horm(MAX_HORM_LEFT);
		ants[i].set_state(-1);
		set_seed = i+offset;
		ants[i].set_seed(set_seed);
		ants[i].ini_prefer_direction();
	}
    for(int i=0;i<tpoints;++i)
    {
        blue_horm[i] = 0;
        red_horm[i] = 0;
    }
}

int main(int argc, char* argv[])
{
	init();

	unsigned int thread_id[THREAD_COUNT];
	unsigned int thread;
	pthread_t* thread_handles;
	thread_handles = (pthread_t*) malloc(THREAD_COUNT*sizeof(pthread_t));

	for(thread = 0; thread < THREAD_COUNT; thread++){
		thread_id[thread] = thread;
		pthread_create(&thread_handles[thread], NULL, run_ants,(void*) &thread_id[thread]);
	}

	char key = 'w';
	while(key != 'x')
	{
		cin >> key;
	}
	return 0;
}
